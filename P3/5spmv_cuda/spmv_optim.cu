#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"

#define N 1024
#define ROWSIZE 9
#define THREADS_PER_BLOCK 128



__global__ void cuspmv(int m, double* dvals, int *dcols, double* dx, double *dy)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x; // Identificar l'index de la fila

    __shared__ double shared_dx[THREADS_PER_BLOCK * ROWSIZE]; //memoria compartida per dx

    if (row < m) //Entrem si la fila es troba dins del rang de m, es a dir, si la fila es troba dins de la matriu.
    {
        double dot = 0.0;

    
        for (int j = 0; j < ROWSIZE; j++) { //Carreguem els elements necessaris de dx a la memoria compartida
            int col_idx = dcols[row * ROWSIZE + j]; 
            if (col_idx < N) { //Si el valor es troba dins de la matriu
                shared_dx[threadIdx.x * ROWSIZE + j] = dx[col_idx]; //Carreguem el valor de dx a la memoria compartida
            }
        }

        //Syncronitzem els threads per assegurar que tots els threads han carregat els valors a la memoria compartida
        __syncthreads(); 

        for (int j = 0; j < ROWSIZE; j++) { // Calculem el producte escalar
            dot += dvals[row * ROWSIZE + j] * shared_dx[threadIdx.x * ROWSIZE + j]; 
        }
        dy[row] = dot; //Guardem a la memoria global el resultat
    }
}




void spmv_cpu(int m, int r, double* vals, int* cols, double* x, double* y)
{
    for(int i = 0; i < m; i++) {
        y[i] = 0.0;   
        for(int j = 0; j < r; j++){
            y[i] += vals[j + i*r]*x[cols[j + i*r]]; // (j + i*r) calcula l'index del element (si no s'enten fer a paper per veure que si funciona)
        }
    }
}


void fill_matrix(double* vals, int* cols)
{

    int indx[ROWSIZE];
    int row_count = 0;
    for(int j = 0; j < N ; j++){
        for(int i = 0; i < N; i++){

            indx[0] = i     + (j - 2)*N;
            indx[1] = i     + (j - 1)*N;
            indx[2] = i - 2 +  j     *N;
            indx[3] = i - 1 +  j     *N;
            indx[4] = i     +  j     *N;
            indx[5] = i + 1 +  j     *N;
            indx[6] = i + 2 +  j     *N;
            indx[7] = i     + (j + 1)*N;
            indx[8] = i     + (j + 2)*N;

            for(int row = 0; row < ROWSIZE; row++)
            {
                if(indx[row] < 0 || indx[row] >= N*N)
                {
                    cols[row + row_count*ROWSIZE] = i + j*N;
                    vals[row + row_count*ROWSIZE] = 0.0;
                }
                else
                {
                    cols[row + row_count*ROWSIZE] = indx[row];
                    if(row == 4)
                    {
                        vals[row + row_count*ROWSIZE] = 0.95;
                    }
                    else
                    {
                        vals[row + row_count*ROWSIZE] = -0.95/(ROWSIZE - 1);
                    }
                }
            }
            row_count++;
        }
    }

    vals[4 + (N*N/2 + N/2)*ROWSIZE] =  1.001*vals[4 + (N*N/2 + N/2)*ROWSIZE];
}

int main()
{
    int vec_size = N*N;

    float time_cpu, time_gpu;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    double* x     = (double*) malloc (vec_size*sizeof(double));
    double* y_cpu = (double*) malloc (vec_size*sizeof(double));
    double* y_gpu = (double*) malloc (vec_size*sizeof(double));

    double* Avals = (double*) malloc (ROWSIZE*vec_size*sizeof(double));
    int*    Acols = (int   *) malloc (ROWSIZE*vec_size*sizeof(int));


    // fill vector with sinusoidal for testing the code
    for(int i = 0; i < vec_size; i++)
    {
        x[i] = sin(i*0.01);
        y_cpu[i] = 0.0;
    }

    fill_matrix(Avals, Acols);

    // measure time of CPU implementation
    hipEventRecord(start);

    for (int i = 0; i < 100; ++i)
        spmv_cpu(vec_size, ROWSIZE, Avals, Acols, x, y_cpu);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_cpu, start, stop);


    // complete here your cuda code
    double* dx;
    double* dy_gpu;

    double* dAvals;
    int*    dAcols;


    // allocate arrays in GPU
    hipMalloc(&dAvals, ROWSIZE*vec_size*sizeof(double));
    hipMalloc(&dAcols, ROWSIZE*vec_size*sizeof(int));
    hipMalloc(&dx, vec_size*sizeof(double));
    hipMalloc(&dy_gpu, vec_size*sizeof(double));

    // transfer data to GPU
    hipMemcpy(dAvals, Avals, vec_size * ROWSIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dAcols, Acols, vec_size* ROWSIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dx, x, vec_size * sizeof(double), hipMemcpyHostToDevice);

    // calculate threads and blocks
    int threadsPerBlock = THREADS_PER_BLOCK;

    // create the gridBlock
    int blocksPerGrid = (vec_size + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(start);

    for( int i=0; i<100; i++){
        // call your GPU kernel here
        cuspmv<<<blocksPerGrid, threadsPerBlock>>>(vec_size, dAvals, dAcols, dx, dy_gpu); 
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_gpu, start, stop);

    // transfer result to CPU RAM
    hipMemcpy(y_gpu, dy_gpu, vec_size * sizeof(double), hipMemcpyDeviceToHost);

    // free arrays in GPU
    hipFree(dAvals);
    hipFree(dAcols);
    hipFree(dx);
    hipFree(dy_gpu);

    // comparison between gpu and cpu results
    double norm2 = 0.0;
    for(int i = 0; i < vec_size; i++)
        norm2 += (y_cpu[i] - y_gpu[i])*(y_cpu[i] - y_gpu[i]);

    norm2 = sqrt(norm2);

    printf("spmv comparison cpu vs gpu error: %e, size %d\n", norm2, vec_size);

    printf("CPU Time: %lf\n", time_cpu/1000);
    printf("GPU Time: %lf\n", time_gpu/1000);

    // free CPU arrays
    free(x);
    free(y_cpu);
    free(y_gpu);
    free(Acols);
    free(Avals);
}
